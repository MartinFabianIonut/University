#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <chrono>
#include <string>

const std::string OUTPUT_PATH = "C:\\GIT\\University\\Year 3\\Parallel and Distributed Programming\\Laboratory 7\\P2\\Outputs\\parallel.txt";
using namespace std;

int* convolutionMatrix;
int* inputMatrix;
int* outputMatrix; // Adăugat pentru a stoca rezultatele în CUDA
int MAX_N, MAX_M, K, P;

int* allocateMatrix(int n, int m)
{
    return new int[n * m];
}

void deallocateMatrix(int* matrix)
{
    delete[] matrix;
}

void readMatrixFromFile(const std::string& filePath, int* matrix, int n, int m)
{
    std::ifstream fin(filePath);
    int nothing;
    fin >> nothing >> nothing;
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            fin >> matrix[i * m + j];
    fin.close();
}

void writeMatrixToFile(const std::string& filePath, int* matrix, int n, int m)
{
    std::ofstream fout(filePath);
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < m; ++j) {
            fout << matrix[i * m + j] << " ";
        }
        fout << "\n";
    }
    fout.close();
}

__global__ void matrixMultiply(int* inputMatrix, int* convolutionMatrix, int* outputMatrix, int N, int M, int K)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M)
    {
        int result = 0;
        for (int i = 0; i < K; ++i)
        {
            for (int j = 0; j < K; ++j)
            {
                int x = min(max(row - K / 2 + i, 0), N - 1);
                int y = min(max(col - K / 2 + j, 0), M - 1);

                if (x >= 0 && x < N && y >= 0 && y < M)
                {
                    result += inputMatrix[x * M + y] * convolutionMatrix[i * K + j];
                }
            }
        }
        outputMatrix[row * M + col] = result;
    }
}

void startCUDA()
{
    int* deviceInputMatrix;
    int* deviceConvolutionMatrix;
    int* deviceOutputMatrix;

    hipMalloc((void**)&deviceInputMatrix, MAX_N * MAX_M * sizeof(int));
    hipMalloc((void**)&deviceConvolutionMatrix, K * K * sizeof(int));
    hipMalloc((void**)&deviceOutputMatrix, MAX_N * MAX_M * sizeof(int));

    hipMemcpy(deviceInputMatrix, inputMatrix, MAX_N * MAX_M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceConvolutionMatrix, convolutionMatrix, K * K * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16); // Ajustați dimensiunea blocului după necesități
    dim3 gridSize((MAX_M + blockSize.x - 1) / blockSize.x, (MAX_N + blockSize.y - 1) / blockSize.y);

    matrixMultiply << <gridSize, blockSize >> > (deviceInputMatrix, deviceConvolutionMatrix, deviceOutputMatrix, MAX_N, MAX_M, K);

    hipMemcpy(outputMatrix, deviceOutputMatrix, MAX_N * MAX_M * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceInputMatrix);
    hipFree(deviceConvolutionMatrix);
    hipFree(deviceOutputMatrix);
}

int main(int argc, char* argv[])
{
    P = 4;
    MAX_N = 1000;
    MAX_M = 1000;
    K = 5;

    string INPUT_PATH = "C:\\GIT\\University\\Year 3\\Parallel and Distributed Programming\\Laboratory 7\\P2\\Inputs\\data" + to_string(MAX_N) + "x" + to_string(MAX_M) + ".txt";
    string CONVOLUTION_PATH = "C:\\GIT\\University\\Year 3\\Parallel and Distributed Programming\\Laboratory 7\\P2\\Inputs\\convolution" + to_string(K) + "x" + to_string(K) + ".txt";
    ifstream fin(CONVOLUTION_PATH);
    fin >> K >> K;
    fin.close();
    convolutionMatrix = allocateMatrix(K, K);

    ifstream fin2(INPUT_PATH);
    fin2 >> MAX_N >> MAX_M;
    fin2.close();

    inputMatrix = allocateMatrix(MAX_N, MAX_M);
    // Allocați și inițializați matricea de ieșire pentru CUDA
    outputMatrix = allocateMatrix(MAX_N, MAX_M);

    auto startTime = std::chrono::high_resolution_clock::now();

    readMatrixFromFile(CONVOLUTION_PATH, convolutionMatrix, K, K);
    readMatrixFromFile(INPUT_PATH, inputMatrix, MAX_N, MAX_M);

    startCUDA();

    auto endTime = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration<double, std::milli>(endTime - startTime).count();
    std::string durationString = std::to_string(duration);

    writeMatrixToFile(OUTPUT_PATH, outputMatrix, MAX_N, MAX_M);

    std::cout << durationString;

    deallocateMatrix(convolutionMatrix);
    deallocateMatrix(inputMatrix);
    deallocateMatrix(outputMatrix);

    return 0;
}
