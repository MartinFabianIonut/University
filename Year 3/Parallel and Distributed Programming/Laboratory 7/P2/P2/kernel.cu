#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <chrono>
#include <string>

using namespace std;
const string OUTPUT_PATH = "C:\\GIT\\University\\Year 3\\Parallel and Distributed Programming\\Laboratory 7\\P2\\Outputs\\parallel.txt";

int* convolutionMatrix;
int* inputMatrix;
int* outputMatrix;
int MAX_N, MAX_M, K, P;

int* allocateMatrix(int n, int m)
{
	return new int[n * m];
}

void deallocateMatrix(int* matrix)
{
	delete[] matrix;
}

void readMatrixFromFile(const string& filePath, int* matrix, int n, int m)
{
	ifstream fin(filePath);
	int nothing;
	fin >> nothing >> nothing;
	for (int i = 0; i < n; ++i)
		for (int j = 0; j < m; ++j)
			fin >> matrix[i * m + j];
	fin.close();
}

void writeMatrixToFile(const string& filePath, int* matrix, int n, int m)
{
	ofstream fout(filePath);
	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < m; ++j) {
			fout << matrix[i * m + j] << " ";
		}
		fout << "\n";
	}
	fout.close();
}

__global__ void matrixMultiply(int* inputMatrix, int* convolutionMatrix, int* outputMatrix, int N, int M, int K)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < N && col < M)
	{
		int result = 0;
		for (int i = 0; i < K; ++i)
		{
			for (int j = 0; j < K; ++j)
			{
				int x = min(max(row - K / 2 + i, 0), N - 1);
				int y = min(max(col - K / 2 + j, 0), M - 1);

				if (x >= 0 && x < N && y >= 0 && y < M)
				{
					result += inputMatrix[x * M + y] * convolutionMatrix[i * K + j];
				}
			}
		}
		outputMatrix[row * M + col] = result;
	}
}

void startCUDA()
{
	int* deviceInputMatrix;
	int* deviceConvolutionMatrix;
	int* deviceOutputMatrix;

	hipMalloc((void**)&deviceInputMatrix, MAX_N * MAX_M * sizeof(int));
	hipMalloc((void**)&deviceConvolutionMatrix, K * K * sizeof(int));
	hipMalloc((void**)&deviceOutputMatrix, MAX_N * MAX_M * sizeof(int));

	hipMemcpy(deviceInputMatrix, inputMatrix, MAX_N * MAX_M * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceConvolutionMatrix, convolutionMatrix, K * K * sizeof(int), hipMemcpyHostToDevice);

	dim3 blockSize(P, P);
	dim3 gridSize((MAX_M + blockSize.x - 1) / blockSize.x, (MAX_N + blockSize.y - 1) / blockSize.y);

	matrixMultiply << <gridSize, blockSize >> > (deviceInputMatrix, deviceConvolutionMatrix, deviceOutputMatrix, MAX_N, MAX_M, K);

	hipMemcpy(outputMatrix, deviceOutputMatrix, MAX_N * MAX_M * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceInputMatrix);
	hipFree(deviceConvolutionMatrix);
	hipFree(deviceOutputMatrix);
}

int main(int argc, char* argv[])
{

	if (argc < 4)
	{
		cout << "Usage: ./your_program <No_of_threads> <MAX_N> <MAX_M> <K>\n";
	}

	P = stoi(argv[1]);
	MAX_N = stoi(argv[2]);
	MAX_M = stoi(argv[3]);
	K = stoi(argv[4]);
	/*
	P = 44;
	MAX_N = 10;
	MAX_M = 10;
	K = 3;
	*/

	string INPUT_PATH = "C:\\GIT\\University\\Year 3\\Parallel and Distributed Programming\\Laboratory 7\\P2\\Inputs\\data" + to_string(MAX_N) + "x" + to_string(MAX_M) + ".txt";
	string CONVOLUTION_PATH = "C:\\GIT\\University\\Year 3\\Parallel and Distributed Programming\\Laboratory 7\\P2\\Inputs\\convolution" + to_string(K) + "x" + to_string(K) + ".txt";
	ifstream fin(CONVOLUTION_PATH);
	fin >> K >> K;
	fin.close();
	convolutionMatrix = allocateMatrix(K, K);

	ifstream fin2(INPUT_PATH);
	fin2 >> MAX_N >> MAX_M;
	fin2.close();

	inputMatrix = allocateMatrix(MAX_N, MAX_M);
	outputMatrix = allocateMatrix(MAX_N, MAX_M);

	auto startTime = chrono::high_resolution_clock::now();

	readMatrixFromFile(CONVOLUTION_PATH, convolutionMatrix, K, K);
	readMatrixFromFile(INPUT_PATH, inputMatrix, MAX_N, MAX_M);

	startCUDA();

	auto endTime = chrono::high_resolution_clock::now();

	auto duration = chrono::duration<double, milli>(endTime - startTime).count();
	string durationString = to_string(duration);

	writeMatrixToFile(OUTPUT_PATH, outputMatrix, MAX_N, MAX_M);

	cout << durationString;

	deallocateMatrix(convolutionMatrix);
	deallocateMatrix(inputMatrix);
	deallocateMatrix(outputMatrix);

	return 0;
}
